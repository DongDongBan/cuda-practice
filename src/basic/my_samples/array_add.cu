#include "hip/hip_runtime.h"
#include <stdio.h>
#include "error.h"

__global__ void intAdd(const int *a, const int *b, int *c) {
  int tx = threadIdx.x;
  c[tx] = a[tx] + b[tx];
}

__host__ void gpuIntAdd(const int a, const int b, int *c, const unsigned int len) {
  int *dev_a, *dev_b, *dev_c; // device copies of a, b, c

  const unsigned int size = sizeof(int) * len; // bytes for and integer

  // allocate device copies of a, b, c
  HANDLE_ERROR(hipMalloc((void**)&dev_a, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, size));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, size));

  // copy inputs to device
//   HANDLE_ERROR(hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice));
//   HANDLE_ERROR(hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice));

  // launch kernel intAdd()
  intAdd<<< 1, len >>>(dev_a, dev_b, dev_c);

  // copy device result back to host copy of c
  HANDLE_ERROR(hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost));

  // free device
  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
}