#include "reduce.cuh"
#include <thrust/device_vector.h>
#include <cstdio>


extern __host__ real reduce8(real *d_x, const int N) {
    thrust::device_ptr<real> dev_ptr(d_x);

    hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start, 0);
    real ans = thrust::reduce(dev_ptr, dev_ptr+N);
    hipEventRecord(stop, 0); hipEventSynchronize(stop);
    float elapsedTime; hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel with input size %d consumes %f ms.\n", N, elapsedTime);

    return ans;
}